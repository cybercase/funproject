#include "hip/hip_runtime.h"
// Copyright 2012 - Stefano Brilli : stefanobrilli@gmail.com
//
// Computes the Levenshtein distance between 2 files.
//
// Information about the Levenshtein distance: 
// http://en.wikipedia.org/wiki/Levenshtein_distance
//
// Other implementations
// http://en.wikibooks.org/wiki/Algorithm_Implementation/Strings/Levenshtein_distance
//
// sample usage:
// $ ./lev_distance file1 file2
//
// Operations: O(m*n)
// Memory: O(max(m, n))
// where m and n are respectively the length of file1 and file2.
//

#include <cstdlib>
#include <cstdio>
#include <algorithm>
#include <vector>

// Threads per block: since this is just an exercise, we use a fixed
// TPB size. In a real case the best configuration should be searched
// based on the underlying hardware

#define TPB 128

// A simple macro to check the return value from the CUDA calls.
// A better macro could be the CUDA_SAFE_CALL from the cuda SDK
#define CUDA_CHECK(function_call) if (function_call != hipSuccess) \
{ fprintf(stderr, "ERR: %s\n", #function_call); exit(1); }

// Opens a file_name, allocates a buffer big as the content of the entire file
// and finally put the file content into the buffer.
//
// Returns the file size
// The ownership of the buffer is passed to the caller function.
// YOU MUST CALL free() ON RETURNED BUFFER
//
// No NULLs are not allowed as input
int LoadFileOrDie(const char* file_name, char** file_content) {
    FILE* fd;
    int size = 0;
    bool fail = (fd=fopen(file_name, "rb")) == NULL ||
        fseek(fd, 0, SEEK_END) < 0 ||
        (size = ftell(fd)) < 0 ||
        (*file_content=(char*)malloc(size)) == NULL ||
        fseek(fd, 0, SEEK_SET) != 0 ||
        int(fread(*file_content, 1, size, fd)) != size ||
        fclose(fd) == -1;
    if (fail) {
        fprintf(stderr, "Can't load file: %s\n", file_name);
        exit(1);
    }
    return size;
}

// Returns the Levenshtein distance to change file1_data into file2_data.
int LevDistance(char* file1_data, int file1_size,
             char* file2_data, int file2_size) {
    int sz = file2_size+1;
    std::vector<int> current(sz);
    std::vector<int> previous(sz);
    for (int i=0; i < sz; ++i) {
        previous[i] = i;
    }

    for (int i=0; i < file1_size; ++i) {
        current[0] = i+1;
        for (int j=1 ; j < sz; ++j) {
            current[j] = std::min( std::min(previous[j], current[j-1])+1, 
                previous[j-1]+(file1_data[i] != file2_data[j-1] ? 1 : 0));
        }
        std::swap(current, previous);
    }
    return previous[sz-1];
}

// Handles diagonals from 1 to m+1
__global__ void stage0(char* file1_data, char* file2_data, int iteration, 
                        int* curr, int* prev, int* prev2) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx > iteration) {
        return;
    }
    if (idx == 0 || idx == iteration) {
        curr[idx] = iteration;
        return;
    }
    curr[idx] = min( min(prev[idx-1], prev[idx])+1, 
        prev2[idx-1] + int(file1_data[iteration-idx-1] != file2_data[idx-1]));
}

// Handles diagonals from m+2 to n+1
template <int ALIGN>
__global__ void stage1(char* file1_data, char* file2_data, int len, int iteration,
                        int* curr, int* prev, int* prev2) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx > len) {
        return;
    }
    if (idx == len) {
        curr[len] = iteration+len+1;
        return;
    }
    curr[idx] = min( min(prev[idx], prev[idx+1])+1, 
        prev2[idx+ALIGN] + int(file1_data[len-idx-1] != file2_data[idx+iteration]));
}

// Handles diagonals from n+2 to n+m
template <int ALIGN>
__global__ void stage2(char* file1_data, char* file2_data, int len, int iteration, int gap,
                        int* curr, int* prev, int* prev2) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= len-iteration) {
        return;
    }
    curr[idx] = min( min(prev[idx], prev[idx+1])+1, 
        prev2[idx+ALIGN] + int(file1_data[len-idx-1] != file2_data[iteration+idx+gap]));
}

// Returns the Levenshtein distance to change file1_data into file2_data.
// 
// WARNING: the function assumes that file2_size >= file1_size
//
// Information about this function can be found at http:://cybcode.blogspot.com
// inside the series of posts "A GPU Exercise"
int CudaLevDistance(char* file1_data, // X
             int file1_size, // 
             char* file2_data,
             int file2_size) {
    int sz = file1_size+1;
    int gap = file2_size-file1_size;

    // GPU pointers
    char* d_file1_data;
    char* d_file2_data;
    int* d_current;
    int* d_previous;
    int* d_previous2;

    // Allocates GPU memory
    CUDA_CHECK( hipMalloc(&d_current, sz*sizeof(int)) ); // Current diagonal
    CUDA_CHECK( hipMalloc(&d_previous, sz*sizeof(int)) ); // Previous diagonal
    CUDA_CHECK( hipMalloc(&d_previous2, sz*sizeof(int)) ); // Previous of previous diagonal
    CUDA_CHECK( hipMalloc(&d_file1_data, file1_size)); // file1_data on GPU
    CUDA_CHECK( hipMalloc(&d_file2_data, file2_size)); // file2_data on GPU

    // Inits values
    CUDA_CHECK( hipMemcpy(d_file1_data, file1_data, file1_size, hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemcpy(d_file2_data, file2_data, file2_size, hipMemcpyHostToDevice));
    CUDA_CHECK( hipMemset(d_previous, 0, sizeof(int)*sz) );
    CUDA_CHECK( hipMemset(d_current, 0, sizeof(int)*sz) );
    CUDA_CHECK( hipMemset(d_previous2, 0, sizeof(int)*sz) );

    // Step 1: diagonals from 1 to m+1
    for (int i=1; i < file1_size+1; ++i) {
        stage0<<<(file1_size+TPB-1)/TPB+1, TPB, TPB*sizeof(int)>>>(d_file1_data, d_file2_data, i, 
            d_current, d_previous, d_previous2);
        std::swap(d_current, d_previous2);
        std::swap(d_previous2, d_previous);
    }

    // Step 2: diagonals from m+2 to n+1
    if ( gap > 0 ) { // Special case
        stage1 <0> <<<(file1_size+TPB-1)/TPB+1, TPB>>>(d_file1_data, d_file2_data,
            file1_size, 0, d_current, d_previous, d_previous2);
        std::swap(d_current, d_previous2);
        std::swap(d_previous2, d_previous);
    }
    for (int i=1; i < gap; ++i) {
        stage1 <1> <<<(file1_size+TPB-1)/TPB+1, TPB>>>(d_file1_data, d_file2_data,
            file1_size, i, d_current, d_previous, d_previous2);
        std::swap(d_current, d_previous2);
        std::swap(d_previous2, d_previous);
    }

    // Step 3: diagonals from n+2 to n+m
    if ( gap == 0 ) {
        stage2 <0> <<<(file1_size+TPB-1)/TPB+1, TPB>>>(d_file1_data, d_file2_data,
            file1_size, 0, gap, d_current, d_previous, d_previous2);
    } else {
        stage2 <1> <<<(file1_size+TPB-1)/TPB+1, TPB>>>(d_file1_data, d_file2_data,
            file1_size, 0, gap, d_current, d_previous, d_previous2);
    }
    std::swap(d_current, d_previous2);
    std::swap(d_previous2, d_previous);

    for (int i=1; i < file1_size; ++i) {
        stage2 <1> <<<(file1_size+TPB-1)/TPB+1, TPB>>>(d_file1_data, d_file2_data,
            file1_size, i, gap, d_current, d_previous, d_previous2);
        std::swap(d_current, d_previous2);
        std::swap(d_previous2, d_previous);
    }

    // Fetch the result from last diagonal (of lenght 1)
    int result = -1;
    hipMemcpy(&result, d_previous, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_current);
    hipFree(d_previous);
    hipFree(d_previous2);
    hipFree(d_file1_data);
    hipFree(d_file2_data);
    return result;
}

int main(int argc, char** argv) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s file1 file2\n", argv[0]);
        return 1;
    }

    char* file1 = NULL;
    int file1_size = LoadFileOrDie(argv[1], &file1);

    char* file2 = NULL;
    int file2_size = LoadFileOrDie(argv[2], &file2);

    printf("%s size is %d bytes\n", argv[1], file1_size);
    printf("%s size is %d bytes\n", argv[2], file2_size);

    if (file1_size > file2_size) {
        std::swap(file1_size, file2_size);
        std::swap(file1, file2);
    }

    clock_t timer = clock();
    printf("LevDistance...\n");
    int ld = LevDistance(file1, file1_size, file2, file2_size);
    printf("elapsed time: %.3f (s)\n", double(clock()-timer)/CLOCKS_PER_SEC);

    timer = clock();
    printf("CudaLevDistance...\n");
    int cld = CudaLevDistance(file1, file1_size, file2, file2_size);
    printf("elapsed time: %.3f (s)\n", double(clock()-timer)/CLOCKS_PER_SEC);

    if (ld != cld) {
        fprintf(stderr, "CRITICAL: LevDistance result differs from CudaLevDistance!");
        exit(1);
    } else {
        printf("OK: CPU-GPU Result matches!\n");
    }

    printf("Distance: %d\n", cld);

    // No need to free memory since the program ends
    free(file1);
    free(file2);

    return 0;
}
